
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <fstream>
#include <chrono>

using namespace std;
using namespace std::chrono;

#define  MAX_NUM_OF_ROWS 1000
#define	 MAX_NUM_OF_COLS 1000

float Matrix_1[MAX_NUM_OF_ROWS][MAX_NUM_OF_COLS];
float Matrix_2[MAX_NUM_OF_ROWS][MAX_NUM_OF_COLS];
float Matrix_output[MAX_NUM_OF_ROWS][MAX_NUM_OF_COLS];

int no_of_rows_1 = 64;
int no_of_rows_2 = 64;
int	no_of_cols_1 = 64;
int	no_of_cols_2 = 64;

void ReadMatrix_1_2(void);
hipError_t MultiplyWithCuda(void);

__global__ void MultiplyKernel(int *c, const int *a, const int *b)
{
    
}

int main()
{
   
	// read 2 Matrix from Files
	ReadMatrix_1_2();
	// take time snap before multiplication
	high_resolution_clock::time_point t1 = high_resolution_clock::now();
	//CPU Multiplication MAtrix_1 * Matrix_2
    hipError_t cudaStatus = MultiplyWithCuda();
	// take time snap after multiplication
	high_resolution_clock::time_point t2 = high_resolution_clock::now();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
	
	// print the Time taken to Multiply two Matrices 
	auto duration = duration_cast<microseconds>(t2 - t1).count();
	cout << "Multiplication Time CPU(us):" << duration << "\n";

	//print the output matrix for testing
//	for (int i = 0; i < no_of_cols_1; i++) {
//		for (int j = 0; j < no_of_rows_2; j++)
//		{
//			cout << Matrix_output[i][j] << "	";
//		}
//		cout << "\n";
//	}

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// this function read the two matrices from two files with the dimensions given in the global variables set at the begining of the file
void ReadMatrix_1_2(void) {
	int x, y;
	ifstream in_1("Matrix_1.txt", std::ifstream::in);
	ifstream in_2("Matrix_2.txt", std::ifstream::in);

	if (!in_1 || !in_2) {
		cout << "Error! Cannot open file.\n";
		return;
	}
	else if (no_of_cols_1 != no_of_rows_2) {
		cout << "Error! Matrix dimensions is not valid for multiplication.\n";
		return;
	}

	for (y = 0; y < no_of_cols_1; y++) {
		for (x = 0; x < no_of_rows_1; x++) {
			in_1 >> Matrix_1[x][y];
		}
	}
	for (y = 0; y < no_of_cols_2; y++) {
		for (x = 0; x < no_of_rows_2; x++) {
			in_2 >> Matrix_2[x][y];
		}
	}

	in_1.close();
	in_2.close();
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t MultiplyWithCuda()
{
    int *dev_a = 0; //Matrix_1
    int *dev_b = 0; //Matrix_2
    int *dev_c = 0; //Matrix_output
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, no_of_rows_1*no_of_cols_2* sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, no_of_rows_1*no_of_cols_1 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, no_of_rows_2*no_of_cols_2 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, Matrix_1, no_of_rows_1*no_of_cols_1 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, Matrix_2, no_of_rows_2*no_of_cols_2 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
	MultiplyKernel <<<1, 5 >>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(Matrix_output, dev_c, no_of_rows_1*no_of_cols_2 * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
